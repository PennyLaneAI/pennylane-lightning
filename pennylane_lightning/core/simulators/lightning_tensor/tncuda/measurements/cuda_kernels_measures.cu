#include "hip/hip_runtime.h"
// Copyright 2024 Xanadu Quantum Technologies Inc.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//     http://www.apache.org/licenses/LICENSE-2.0
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/**
 * @file cuda_kernels_measures.cu
 */
#include <hip/hip_complex.h>

#include "cuError.hpp"
#include "cuda_helpers.hpp"

namespace Pennylane::LightningTensor::TNCuda::Measures {

/**
 * @brief The CUDA kernel that calculates the probability from a given state
 * tensor data on GPU device.
 *
 * @tparam GPUDataT hipComplex data type (hipComplex or hipDoubleComplex).
 * @tparam PrecisionT Floating data type.
 *
 * @param state Complex data pointer of state tensor on device.
 * @param probs The probability result on device.
 * @param data_size The length of state tensor on device.
 */
template <class GPUDataT, class PrecisionT>
__global__ void getProbsKernel(GPUDataT *state, PrecisionT *probs,
                               const int data_size) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < data_size) {
        PrecisionT real = state[i].x;
        PrecisionT imag = state[i].y;
        probs[i] = real * real + imag * imag;
    }
}

/**
 * @brief The CUDA kernel that normalize the probability from a given state
 * tensor data on GPU device.
 *
 * @tparam PrecisionT Floating data type.
 *
 * @param probs The probability to be normalized.
 * @param data_size The length of state tensor on device.
 * @param sum The sum of all probabilities.
 */
template <class PrecisionT>
__global__ void normalizeProbsKernel(PrecisionT *probs, const int data_size,
                                     const PrecisionT sum) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < data_size) {
        probs[i] /= sum;
    }
}

/**
 * @brief The CUDA kernel call wrapper.
 *
 * @tparam GPUDataT hipComplex data type (hipComplex or hipDoubleComplex).
 * @tparam PrecisionT Floating data type.
 *
 * @param state Complex data pointer of state tensor on device.
 * @param probs The probability result on device.
 * @param data_size The length of state tensor on device.
 * @param thread_per_block Number of threads set per block.
 * @param stream_id Stream id of CUDA calls
 */
template <class GPUDataT, class PrecisionT>
void getProbs_CUDA_call(GPUDataT *state, PrecisionT *probs, const int data_size,
                        std::size_t thread_per_block, hipStream_t stream_id) {
    auto dv = std::div(data_size, thread_per_block);
    const std::size_t num_blocks = dv.quot + (dv.rem == 0 ? 0 : 1);
    const std::size_t block_per_grid = (num_blocks == 0 ? 1 : num_blocks);
    dim3 blockSize(thread_per_block, 1, 1);
    dim3 gridSize(block_per_grid, 1);

    getProbsKernel<GPUDataT, PrecisionT>
        <<<gridSize, blockSize, 0, stream_id>>>(state, probs, data_size);
    PL_CUDA_IS_SUCCESS(hipGetLastError());
}

/**
 * @brief The CUDA kernel call wrapper.
 *
 * @tparam PrecisionT Floating data type.
 *
 * @param probs The probability to be normalized.
 * @param data_size The length of state tensor on device.
 * @param thread_per_block Number of threads set per block.
 * @param stream_id Stream id of CUDA calls
 */
template <class PrecisionT>
void normalizeProbs_CUDA_call(PrecisionT *probs, const int data_size,
                              const PrecisionT sum,
                              std::size_t thread_per_block,
                              hipStream_t stream_id) {
    auto dv = std::div(data_size, thread_per_block);
    const std::size_t num_blocks = dv.quot + (dv.rem == 0 ? 0 : 1);
    const std::size_t block_per_grid = (num_blocks == 0 ? 1 : num_blocks);
    dim3 blockSize(thread_per_block, 1, 1);
    dim3 gridSize(block_per_grid, 1);

    normalizeProbsKernel<PrecisionT>
        <<<gridSize, blockSize, 0, stream_id>>>(probs, data_size, sum);
    PL_CUDA_IS_SUCCESS(hipGetLastError());
}

// Definitions
/**
 * @brief Explicitly get the probability of given state tensor data on GPU
 * device.
 *
 * @param state Complex data pointer of state tensor on device.
 * @param probs The probability result on device.
 * @param data_size The length of state tensor on device.
 * @param thread_per_block Number of threads set per block.
 * @param stream_id Stream id of CUDA calls
 */
void getProbs_CUDA(hipComplex *state, float *probs, const int data_size,
                   const std::size_t thread_per_block, hipStream_t stream_id) {
    getProbs_CUDA_call<hipComplex, float>(state, probs, data_size,
                                         thread_per_block, stream_id);
}

void getProbs_CUDA(hipDoubleComplex *state, double *probs, const int data_size,
                   const std::size_t thread_per_block, hipStream_t stream_id) {
    getProbs_CUDA_call<hipDoubleComplex, double>(state, probs, data_size,
                                                thread_per_block, stream_id);
}

/**
 * @brief Explicitly get the probability of given state tensor data on GPU
 * device.
 *
 * @param probs The probability to be normalized.
 * @param data_size The length of state tensor on device.
 * @param thread_per_block Number of threads set per block.
 * @param stream_id Stream id of CUDA calls
 */
void normalizeProbs_CUDA(float *probs, const int data_size, const float sum,
                         const std::size_t thread_per_block,
                         hipStream_t stream_id) {
    normalizeProbs_CUDA_call<float>(probs, data_size, sum, thread_per_block,
                                    stream_id);
}

void normalizeProbs_CUDA(double *probs, const int data_size, const double sum,
                         const std::size_t thread_per_block,
                         hipStream_t stream_id) {
    normalizeProbs_CUDA_call<double>(probs, data_size, sum, thread_per_block,
                                     stream_id);
}
} // namespace Pennylane::LightningTensor::TNCuda::Measures
